#include "hip/hip_runtime.h"
/* \file
 * \author Chris Bradley
 * \brief This file provides the routines for solving differential-algebraic equations with an external solver.
 *.
 * \section LICENSE
 * 
 * Version: MPL 1.1/GPL 2.0/LGPL 2.1
 * 
 * The contents of this file are subject to the Mozilla Public License
 * Version 1.1 (the "License"); you may not use this file except in
 * compliance with the License. You may obtain a copy of the License at
 * http://www.mozilla.org/MPL/
 * 
 * Software distributed under the License is distributed on an "AS IS"
 * basis, WITHOUT WARRANTY OF ANY KIND, either express or implied. See the
 * License for the specific language governing rights and limitations
 * under the License.
 * 
 * The Original Code is OpenCMISS
 * 
 * The Initial Developer of the Original Code is University of Auckland,
 * Auckland, New Zealand, the University of Oxford, Oxford, United
 * Kingdom and King's College, London, United Kingdom. Portions created
 * by the University of Auckland, the University of Oxford and King's
 * College, London are Copyright (C) 2007-2010 by the University of
 * Auckland, the University of Oxford and King's College, London.
 * All Rights Reserved.
 * 
 * Contributor(s):
 * 
 * Alternatively, the contents of this file may be used under the terms of
 * either the GNU General Public License Version 2 or later (the "GPL"), or
 * the GNU Lesser General Public License Version 2.1 or later (the "LGPL"),
 * in which case the provisions of the GPL or the LGPL are applicable instead
 * of those above. If you wish to allow use of your version of this file only
 * under the terms of either the GPL or the LGPL, and not to allow others to
 * use your version of this file under the terms of the MPL, indicate your
 * decision by deleting the provisions above and replace them with the notice
 * and other provisions required by the GPL or the LGPL. If you do not delete
 * the provisions above, a recipient may use your version of this file under
 * the terms of any one of the MPL, the GPL or the LGPL.
 * 
 */

/*
File: external_dae_solver_routines.c
===================
 
This file provides provides the routines for solving differential-algebraic equations with an external solver.

Functions included:

SolverDAEExternalIntegrate     Solves the differential-algebraic equation.

*/

/* Included files */
#include <stdio.h>
#include <stdlib.h>

#include "external_dae_solver_routines.h"
#include "cuda_solver_routines.cu"

/* Type definitions */

/* Function Definitions */
extern "C"
{
void SolverDAEExternalIntegrate(const int NumberOfDofs,
				const double StartTime,
				const double EndTime,
				double *InitialStep,
				const int ThreadsPerBlock,
				const int NumberOfPartitions,
				const int NumberOfStreams,
				const int OnlyOneModelIndex,
				int *ModelsData,
				int NumberOfState,
				double *StateData,
				int NumberOfParameters,
				double *ParametersData,
				int NumberOfIntermediate,
				double *IntermediateData,
				int *err)
{
	FILE* timing_file = NULL;
	char *filename = NULL;

	asprintf(&filename,"Results/MonodomainExample-CUDAON-%d-%d-%d-%d.txt",(NumberOfDofs/101) - 1,ThreadsPerBlock,NumberOfPartitions,NumberOfStreams);
	timing_file = fopen(filename, "rt");

	if (!timing_file) {
		timing_file = fopen(filename, "wt");
		fprintf(timing_file,"Cell Model\tIntegrator\tNumber of Threads\tNumber 0f Blocks\tThreads Per Block\tNumber of Partitions\tNumber of Streams\tTotal Computational Time(s)\tTotal GFLOPS\n");
//		fprintf(timing_file,"Cell Model\tIntegrator\tNumber of Threads\tNumber 0f Blocks\tThreads Per Block\tNumber of Partitions\tNumber of Streams\tTotal Computational Time(s)\tTotal GFLOPS\tSingle Kernel Computaional Time(s)\tKernel GFLOPS\tDevice Utilisation\n");
	} else {
		fclose(timing_file);
		timing_file = fopen(filename, "at");
		if (!timing_file) {
			fprintf(stderr, "Timing file could not be opened or created.");
			exit(EXIT_FAILURE);
		}
	}

	//printf("start %f end %f steps %d\n", StartTime, EndTime, (int)((EndTime-StartTime)/InitialStep[0]));
    //  timeSteps = (int)ceil(((EndTime-StartTime)/InitialStep[0]));

	solve(StateData, StartTime, EndTime, InitialStep[0], NumberOfDofs, ThreadsPerBlock, NumberOfPartitions, NumberOfStreams, timing_file);

	if (timing_file != NULL ) fclose(timing_file);
	free(filename);
}
}

