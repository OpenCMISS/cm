#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <cutil_inline.h>

//extern __shared__ double shared_array[];
//const int algebraicCount = 25;
//const int rateStateCount = 8;
//const int constantCount = 0;
//const double FLOPSPerFunction = 193.0f;
//const int DEFAULT_TESTING_THREADS = 2000000;
//const int sharedMemoryCellModel = 0;
//const char* cellModelName = "LR R3";
//
//////////////////////////////////////////////////////////////////////////////////
//// Cell Model Device Functions
//////////////////////////////////////////////////////////////////////////////////
//__device__ void computeRates(float VOI, double* DUMMY, double* STATES, double* ALGEBRAIC, double* RATES)
//{
//	ALGEBRAIC[0] = -25.5; // Add stimulus in proper
//	ALGEBRAIC[1] = (0.32f*STATES[0]+15.0816f)/(1.0f - (expf(- 0.1f*STATES[0]-4.713f))); // 7 ops
//
//	if (STATES[0] < -40.0f) {
//		ALGEBRAIC[2] = 0.135f*(expf(((80.0f+STATES[0])/- 6.8f))); // 4 ops
//		ALGEBRAIC[3] = (( - 127140*(expf(0.2444f*STATES[0])) - 3.47400e-05*(expf(-0.04391f*STATES[0])))*(STATES[0]+37.78f))/(1.0f+(expf(0.311f*STATES[0]+24.64053))); // 14 ops
//		ALGEBRAIC[9] = 3.56f*(expf(0.079f*STATES[0]))+ 310000*(expf(0.35f*STATES[0]));  // 7 ops
//		ALGEBRAIC[10] = 0.1212f*(expf(-0.01052f*STATES[0]))/(1.0f+(expf(-0.1378f*STATES[0]-5.531292f))); // 8 ops
//	} else {
//		ALGEBRAIC[2] = 0.00000;
//		ALGEBRAIC[3] = 0.00000;
//		ALGEBRAIC[9] = 1.0f/( 0.13f*(1.0f+(expf(((STATES[0]+10.66f)/- 11.1f)))));
//		ALGEBRAIC[10] = ( 0.3f*(expf(-2.53500e-07*STATES[0])))/(1.0f+(expf(-0.1f*STATES[0]-3.2f)));
//	}
//	if (STATES[0] < -100.0f) {
//		ALGEBRAIC[16] = 2.837f*(expf(0.04f*STATES[0]+3.08f) - 1.0f)/((STATES[0]+77.0f)*(expf(0.04f*STATES[0]+1.4f))); // 11 ops
//	} else {
//		ALGEBRAIC[16] = 1.0f;
//	}
//
//	ALGEBRAIC[4] = (0.095f*(expf(-0.01f*STATES[0] + 0.5f)))/(1.0f+(expf(-0.072*STATES[0] + 0.36f))); // 9 ops
//	ALGEBRAIC[5] = (0.012f*(expf(-0.008f*STATES[0]-0.224f)))/(1.0f+(expf(0.15f*STATES[0]+4.2f))); // 9 ops
//	ALGEBRAIC[6] = (0.0005f*(expf(0.083f*STATES[0]+4.15f)))/(1.0f+(expf(0.057f*STATES[0]+2.85f))); // 9 ops
//	ALGEBRAIC[7] =  23*(powf(STATES[1], 3.0f))*STATES[2]*STATES[3]*(STATES[0] - 54.794463f); // 6 ops
//	ALGEBRAIC[8] =  0.08f*(expf(-STATES[0]/11.0000)); // 3 ops
//	ALGEBRAIC[11] = (0.07f*(expf(-0.017f*STATES[0]-0.748f)))/(1.0f+(expf(0.05f*STATES[0]+2.2f))); // 9 ops
//	ALGEBRAIC[12] = (0.0065f*(expf(-0.02f*STATES[0]-0.6f)))/(1.0f+(expf(-0.2f*STATES[0]-6.0f))); // 9 ops
//	ALGEBRAIC[13] = (0.0013f*(expf(-0.06f*STATES[0]-1.2f)))/(1.0f+(expf(-0.04f*STATES[0]-0.8f))); // 9 ops
//	ALGEBRAIC[14] = 7.7f - 13.0287f*logf(STATES[4]); // 3 ops
//	ALGEBRAIC[15] =  0.09f*STATES[5]*STATES[6]*(STATES[0] - ALGEBRAIC[14]); // 4 ops
//	ALGEBRAIC[17] =  0.282f*STATES[7]*ALGEBRAIC[16]*(STATES[0] + 77.56758f); // 4 ops
//	ALGEBRAIC[18] = 1.02f/(1.0f+(expf(0.2385f*STATES[0] + 6.83967915f))); // 4 ops
//	ALGEBRAIC[19] = (0.49124f*(expf( 0.08032f *STATES[0] + 7.49939f) + expf(0.06175f*STATES[0] - 31.271255925f)))/(1.00000+expf(-0.514300*STATES[0] - 214.85137268791f)); // 13 ops
//	ALGEBRAIC[20] = ALGEBRAIC[18]/(ALGEBRAIC[18]+ALGEBRAIC[19]); // 2 ops
//	ALGEBRAIC[21] =  0.6047f*ALGEBRAIC[20]*(STATES[0] + 87.89290f); // 3 ops
//	ALGEBRAIC[22] = 1.0f/(1.0f+(exp(((7.488f - STATES[0])/5.98f)))); // 5 ops
//	ALGEBRAIC[23] =  0.0183f*ALGEBRAIC[22]*(STATES[0] + 87.89290f); // 3 ops
//	ALGEBRAIC[24] =  0.03921f*STATES[0] +2.3475027f; // 3 ops
//
//	RATES[0] =  -(ALGEBRAIC[0]+ALGEBRAIC[7]+ALGEBRAIC[15]+ALGEBRAIC[17]+ALGEBRAIC[21]+ALGEBRAIC[23]+ALGEBRAIC[24]); // 7 ops
//	RATES[1] =  ALGEBRAIC[1]*(1.00000 - STATES[1]) -  ALGEBRAIC[8]*STATES[1]; // 4 ops
//	RATES[2] =  ALGEBRAIC[2]*(1.00000 - STATES[2]) -  ALGEBRAIC[9]*STATES[2]; // 4 ops
//	RATES[3] =  ALGEBRAIC[3]*(1.00000 - STATES[3]) -  ALGEBRAIC[10]*STATES[3]; // 4 ops
//	RATES[4] =  - 0.0001f*ALGEBRAIC[15]+ 0.000007f - 0.07f*STATES[4]; // 4 ops
//	RATES[5] =  ALGEBRAIC[4]*(1.00000 - STATES[5]) -  ALGEBRAIC[11]*STATES[5]; // 4 ops
//	RATES[6] =  ALGEBRAIC[5]*(1.00000 - STATES[6]) -  ALGEBRAIC[12]*STATES[6]; // 4 ops
//	RATES[7] =  ALGEBRAIC[6]*(1.00000 - STATES[7]) -  ALGEBRAIC[13]*STATES[7]; // 4 ops
//}

extern __shared__ double shared_array[];
const int rateStateCount = 2;
const int constantCount = 0;
const int algebraicCount = 1;
const double FLOPSPerFunction = 9.0f;
const int DEFAULT_TESTING_THREADS = 20000000;
const int sharedMemoryCellModel = 0;
const char* cellModelName = "FN R1";

////////////////////////////////////////////////////////////////////////////////
// Cell Model Device Functions
////////////////////////////////////////////////////////////////////////////////
__device__ void computeRates(double time, double* constants, double* states, double* algebraic, double* rates)
{
	rates[1] =  0.005f*(states[0] - 3.0f*states[1]);
	rates[0] =  ((states[0]*(states[0] - -0.08f)*(1.0f - states[0]) - states[1])+ algebraic[0]);
}

void initProblem(int num_threads, double* STATES) {
	printf("test\n\n\n\n\n\n");
}

//////////////////////////////////////////////////////////////////////////////////
//// Cell Model Host Functions ////////// Should Not Be Needed Later /////////////
//////////////////////////////////////////////////////////////////////////////////
//void initProblem(int num_threads, double* STATES)
//{
//	int i;
//
//	STATES[0] = -84.3801107371;
//	STATES[1] = 0.00171338077730188;
//	STATES[2] = 0.982660523699656;
//	STATES[3] = 0.989108212766685;
//	STATES[4] = 0.00017948816388306;
//	STATES[5] = 0.00302126301779861;
//	STATES[6] = 0.999967936476325;
//	STATES[7] = 0.0417603108167287;
//
//	for (i=1; i<num_threads; i++)
//		memcpy(STATES + i*rateStateCount, STATES, rateStateCount*sizeof(double));
//}

const int DEFAULT_TESTING_TIMESTEPS = 1000;
const double FLOPSPerTimeStep = 2.0f;
const int FunctionEvals = 1;
const char* integratorName = "E R1";
const int sharedMemoryIntegrator = 0;

////////////////////////////////////////////////////////////////////////////////
// ODE Integrator Device Functions
////////////////////////////////////////////////////////////////////////////////
__device__ void integrator(int timeSteps, float stepSize, double* constants, double* states, double* algebraic)
{
	int i,j;

	double rates[rateStateCount];

#pragma unroll 40
	for (i=1; i<timeSteps+1; i++) {
		computeRates(i*stepSize, constants, states, algebraic, rates);

		for (j=0; j<rateStateCount; j++) {
			states[j] += stepSize*rates[j];
		}
	}
}

//const int DEFAULT_TESTING_TIMESTEPS = 1000;
//const double FLOPSPerTimeStep = 22.0f;
//const int FunctionEvals = 4;
//const char* integratorName = "RK R2";
//const int sharedMemoryIntegrator = 0;
//
//////////////////////////////////////////////////////////////////////////////////
//// ODE Integrator Device Functions
//////////////////////////////////////////////////////////////////////////////////
//__device__ void integrator(int timeSteps, float stepSize, double* constants, double* states, double* algebraic)
//{
//	int i,j;
//
//	double previousStates[rateStateCount];
//	double kutta[rateStateCount];
//	double offsets[rateStateCount];
//
//	for (j=0; j<rateStateCount; j++) {
//		previousStates[j] = states[j];
//	}
//
//
//#pragma unroll 40
//	for (i=1; i<timeSteps+1; i++) {
//		computeRates(i*stepSize, constants, previousStates, algebraic, kutta);
//
//		for (j=0; j<rateStateCount; j++) {
//			offsets[j] = previousStates[j] + 0.5f*kutta[j]*stepSize; // 3 ops
//			states[j] += (stepSize/6)*(kutta[j]); // 3 ops
//		}
//		computeRates(i*stepSize, constants, offsets, algebraic, kutta);
//
//		for (j=0; j<rateStateCount; j++) {
//			offsets[j] = previousStates[j] + 0.5f*kutta[j]*stepSize; // 3 ops
//			states[j] += (stepSize/6)*(2*kutta[j]); // 4 ops
//		}
//		computeRates(i*stepSize, constants, offsets, algebraic, kutta);
//
//		for (j=0; j<rateStateCount; j++) {
//			offsets[j] = previousStates[j] + kutta[j]*stepSize; // 2 ops
//			states[j] += (stepSize/6)*(2*kutta[j]); // 4 ops
//		}
//		computeRates(i*stepSize, constants, offsets, algebraic, kutta);
//
//		for (j=0; j<rateStateCount; j++) {
//			states[j] += (stepSize/6)*(kutta[j]); // 3 ops
//			previousStates[j] = states[j];
//		}
//	}
//}

const int sharedMemoryDevice = rateStateCount + algebraicCount;

////////////////////////////////////////////////////////////////////////////////
// Solver Kernel
////////////////////////////////////////////////////////////////////////////////
__global__ void solveSystem(int timeSteps, float stepSize, double* states)
{
	int i;

	double* threadAlgebraic = &shared_array[algebraicCount*threadIdx.x + sharedMemoryIntegrator*blockDim.x];
	double* threadStates = &shared_array[(algebraicCount + sharedMemoryIntegrator)*blockDim.x];

#if constantCount > 0
	double threadConstants[constantCount];

	intitialiseConstants(threadConstants);
#endif

	for (i=0; i<rateStateCount; i++) {
		threadStates[threadIdx.x + i*blockDim.x] = states[threadIdx.x + i*blockDim.x + blockIdx.x*blockDim.x*rateStateCount];
	}

#if constantCount > 0
	integrator(timeSteps, stepSize, threadConstants, threadStates + threadIdx.x*rateStateCount, threadAlgebraic);
#else
	integrator(timeSteps, stepSize, NULL, threadStates + threadIdx.x*rateStateCount, threadAlgebraic);
#endif

	for (i=0; i<rateStateCount; i++) {
		states[threadIdx.x + i*blockDim.x + blockIdx.x*blockDim.x*rateStateCount] = threadStates[threadIdx.x + i*blockDim.x];
	}
}

////////////////////////////////////////////////////////////////////////////////
// Check Error Checking
////////////////////////////////////////////////////////////////////////////////
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         


}

void *safeMalloc(size_t size)  // Only called once - Pointless??
{
	void *ret = malloc(size);
	if (ret == NULL) {
		fprintf(stderr, "malloc of %zu bytes failed", size);
		exit(EXIT_FAILURE);
	}
	return ret;
}

void cleanup() { hipDeviceReset(); }

void domainDecomposition(unsigned int* threads_per_domain, unsigned int* spill, unsigned int num_threads, int unsigned num_streams, size_t* pagedMemorySize, unsigned int* threads_per_block,
		unsigned int* num_partitions, unsigned int* blocksPerDomain, unsigned int* num_blocks, size_t* sharedMem){

	size_t freeGPUMemory = 0;
	size_t test = 0;
	int cuda_device = 0;
	unsigned int remainingBlocks = 0;
	unsigned int numberRemainingDomains = 0;
	unsigned int count;

    hipDeviceProp_t deviceProp;
    cutilSafeCall( hipGetDeviceProperties(&deviceProp, cuda_device) );

    (*threads_per_block) = (*threads_per_block) > deviceProp.maxThreadsPerBlock ? deviceProp.maxThreadsPerBlock : (*threads_per_block);

	// Adjust threads per blocks so that states variables fit in shared memory
	count=0;
	(*sharedMem) = (size_t) (sharedMemoryIntegrator + sharedMemoryDevice + sharedMemoryCellModel) * (*threads_per_block) * sizeof(double);
	while ( (*sharedMem) > deviceProp.sharedMemPerBlock*0.5 && count < 200 ) {
		if ((*threads_per_block) > 32 ) {
			(*threads_per_block)-=32;
		} else {
			fprintf(stderr, "Cannot fit variables in shared memory");
			exit(EXIT_FAILURE);
		}
		count++;
		(*sharedMem) = (size_t) (sharedMemoryIntegrator + sharedMemoryDevice + sharedMemoryCellModel) * (*threads_per_block) * sizeof(double);
	}
	(*num_blocks)=num_threads/(*threads_per_block);
	spill[2]=num_threads % (*threads_per_block);
	//if (spill[2] !=0) (*num_blocks)++; // Round up calculation

	(*blocksPerDomain) = (*num_blocks)/(num_streams*(*num_partitions));
	if ((*blocksPerDomain) == 0) {
		fprintf(stderr, "Too many streams and partitions for your problem size\nReduce these and try again");
		exit(EXIT_FAILURE);
	} else if ((*blocksPerDomain) > deviceProp.maxGridSize[0]) {
		fprintf(stderr, "Too many blocks allocated to each domain for your problem size\nIncrease number of partitions and try again");
		exit(EXIT_FAILURE);
	}
	remainingBlocks = (*num_blocks)%(num_streams*(*num_partitions));
	if (remainingBlocks > 0 && remainingBlocks <= num_streams*(*blocksPerDomain)) {
		(*num_partitions)++;
	} else if (remainingBlocks > num_streams*(*blocksPerDomain)) {
		numberRemainingDomains = ceil((float)remainingBlocks/(*blocksPerDomain));
		(*num_partitions) += ceil((float)numberRemainingDomains/num_streams);
	}

	if ((*num_blocks)/(num_streams*(*num_partitions))==0) {
		(*num_partitions)--;
	}

	remainingBlocks = (*num_blocks)%(*num_partitions);

	// Adjust number of partitions so that data fits in GPU memory
	count = 0;
	cutilSafeCall( hipMemGetInfo(&freeGPUMemory, &test) );
	*pagedMemorySize = sizeof(double)*rateStateCount*(*blocksPerDomain)*num_streams*(*threads_per_block);
	while ( *pagedMemorySize > freeGPUMemory*0.9 && count < 200 ) {
		if ((*blocksPerDomain) > 1 ) {
			(*num_partitions) = (*num_blocks)/num_streams/--(*blocksPerDomain);
			remainingBlocks = (*num_blocks)%(num_streams*(*num_partitions));
			if (remainingBlocks != 0) {
				(*num_partitions)++;
			} else if (remainingBlocks > num_streams*(*blocksPerDomain)) {
				numberRemainingDomains = ceil((float)remainingBlocks/(*blocksPerDomain));
				(*num_partitions) += ceil((float)numberRemainingDomains/num_streams);
			}
			remainingBlocks = (*num_blocks)%(*num_partitions);
		} else {
			fprintf(stderr, "Cannot fit variables in GPU device memory\nReduce threads per block and try again");
			exit(EXIT_FAILURE);
		}
		count++;

		*pagedMemorySize = sizeof(double)*rateStateCount*(*blocksPerDomain)*num_streams*(*threads_per_block);
	}

	if (*pagedMemorySize > freeGPUMemory*0.9) {
		fprintf(stderr, "Cannot fit variables in GPU device memory\nReduce threads per block and try again");
		exit(EXIT_FAILURE);
	}

	(*threads_per_domain) = (*threads_per_block)*(*blocksPerDomain);
	spill[0] = remainingBlocks/(*blocksPerDomain);
	spill[1] = remainingBlocks%(*blocksPerDomain);

//	if (renainingBlocks != 0) {
//		blocksLastStream =
//	}
//	remainingBlocks = num_threads%(num_blocks*(*num_partitions))
//	for (i=0; i<(*num_partitions); i++) {
//		if (i == (*num_partitions) && remainingBlocks != 0) {
//
//			for (j=0; j<remainingStreams; j++) {
//				domainIndex = j + i*num_streams;
//				(*threads_per_domain) = (*threads_per_block)*(*blocksPerDomain);
//				count += threads_per_domain[domainIndex];
//			}
//			if (blocksLastStream!=0) {
//				domainIndex ++;
//				threads_per_domain[domainIndex] = num_threads - count;
//			}
//		} else {
//			for (j=0; j<num_streams; j++) {
//				domainIndex = j + i*num_streams;
//				threads_per_domain[domainIndex] = (*threads_per_block)*(*blocksPerDomain);
//				count += threads_per_domain[domainIndex];
//			}
//		}
//	}
}

////////////////////////////////////////////////////////////////////////////////
// CPU side solution routine
////////////////////////////////////////////////////////////////////////////////
void solve(double* h_states, double startTime, double endTime, double stepSize,
		   unsigned int num_threads, unsigned int threads_per_block, unsigned int num_partitions, 
		   int unsigned num_streams, FILE *timing_file)
{
    unsigned int i,j;
	unsigned int num_blocks = 0;
	unsigned int local_offset = 0;
	unsigned int global_offset = 0;
	double nFLOPS = 0;
	double dSeconds = 0;
	double gflops = 0;
	double kernel_dSeconds = 0;
	double kernel_nFLOPS = 0;
	double kernel_gflops = 0;
	unsigned int timer = 0;
	unsigned int kernel_timer = 0;
	unsigned int domainIndex = 0;
	unsigned int blocksPerDomain = 0;
	unsigned int lastFullDomain = 0;
	unsigned int threads_per_domain = 0;
	unsigned int spill[3] = { 0 };

	size_t pagedMemorySize;
	size_t sharedMem = 0;
	size_t lastStreamMemorySize = 0;

	double *d_states = 0;
	double *h_paged_states = 0;

	hipStream_t *streams;
	int cuda_device = 0;

	unsigned int timeSteps = (endTime-startTime)/stepSize;

    // Check for a CUDA compatible device
    int num_devices=0;
    cutilSafeCall( hipGetDeviceCount(&num_devices) );
    if(0==num_devices)
    {
        printf("Your system does not have a CUDA capable device\n");
        return;
	}
	
    // Set appropriate device flags and print relevant information
	cutilSafeCall( hipSetDevice( cuda_device ) );
	cutilSafeCall( hipSetDeviceFlags(hipDeviceScheduleBlockingSync) );
    hipDeviceProp_t deviceProp;
    cutilSafeCall( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (1 == deviceProp.major) && (deviceProp.minor < 1))
        printf("%s does not have compute capability 1.1 or later\n", deviceProp.name);

    domainDecomposition(&threads_per_domain, spill, num_threads, num_streams, &pagedMemorySize, &threads_per_block,
    		&num_partitions, &blocksPerDomain, &num_blocks, &sharedMem);


    lastFullDomain = num_blocks/blocksPerDomain;

	// Allocate CUDA device and host pinned memory
	cutilSafeCall( hipHostMalloc((void**) &h_paged_states, pagedMemorySize) );
	cutilSafeCall( hipMalloc((void **) &d_states, pagedMemorySize) );

	// Create streams
	streams = (hipStream_t*) malloc(num_streams * sizeof(hipStream_t));
    for(i = 0; i < num_streams; i++)
        cutilSafeCall( hipStreamCreate(&(streams[i])) );

    // Setup execution parameters
    dim3  grid(blocksPerDomain, 1, 1);
    dim3  threads(threads_per_block, 1, 1);

    //#ifdef DEBUG
    if (timing_file) {
//    	printf("> Device name : %s\n", deviceProp.name );
//		printf("> CUDA Capable SM %d.%d hardware with %d multi-processors\n",
//			deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
//		printf("> Cell Model = %s, Integrator = %s\n", cellModelName, integratorName);
//		printf("> num_threads = %d, num_blocks = %d, threads_per_block = %d, num_partitions = %d, timeSteps = %d, num_streams = %d\n",
//			num_threads, num_blocks, threads_per_block, num_partitions, timeSteps, num_streams);
//		//#endif
//    	printf("grid.x %d threads.x %d sharedMem %d\n", grid.x, threads.x, sharedMem);
//    	printf("Spills %d %d %d\n", spill[0], spill[1], spill[2]);

		// Setup and start global timer
		timer = 0;
		cutCreateTimer(&timer);
		cutilCheckError(cutStartTimer(timer));

//		// Test kernel speed in default stream (timing is more accurate in default stream)
//		memcpy(h_paged_states, h_states, pagedMemorySize/num_streams);
//		cutilSafeCall( hipMemcpy(d_states, h_paged_states, pagedMemorySize/num_streams,
//			hipMemcpyHostToDevice) );
//		// Start kernel timer
//		kernel_timer = 0;
//		cutCreateTimer(&kernel_timer);
//		cutilCheckError(cutStartTimer(kernel_timer));
//		// Start kernel
//		solveSystem<<<grid, threads, sharedMem>>>(timeSteps, stepSize, d_states);
//		checkCUDAError("Single Kernel Execution");
//		cutilSafeCall( hipDeviceSynchronize() );
//		// Stop kernel Timer
//		cutilCheckError(cutStopTimer(kernel_timer));
//		cutilSafeCall( hipMemcpy(h_paged_states, d_states, pagedMemorySize/num_streams,
//			hipMemcpyDeviceToHost) );
//		memcpy(h_states, h_paged_states, pagedMemorySize/num_streams);

		// Prefetch data for next partition in first stream
//		if (num_partitions>1) {
//			global_offset = rateStateCount * num_streams * grid.x * threads.x;
//			memcpy(h_paged_states, h_states + global_offset, pagedMemorySize/num_streams);
//		}
//	} else {
		memcpy(h_paged_states, h_states, pagedMemorySize);
	}

	// Queue kernel calls into streams to hide memory transfers (num_partitions sets of kernel calls in each stream)
	for(i = 0; i < num_partitions+1; i++) {
		// Asynchronously launch num_streams memcopies
		for(j = 0; j < num_streams; j++) { //((timing_file!=NULL && i==0) ? 1 : 0); j < num_streams; j++){
			domainIndex = j + i*num_streams;
			if (domainIndex <= lastFullDomain) {
				local_offset = j * rateStateCount * grid.x * threads.x ;
				if (domainIndex == lastFullDomain && (spill[1]!=0 || spill[2]!=0)) {
					//printf("last async in %d, size %d\n", domainIndex, lastStreamMemorySize);
					cutilSafeCall( hipMemcpyAsync(d_states + local_offset, h_paged_states + local_offset,
							lastStreamMemorySize, hipMemcpyHostToDevice, streams[j]) );
				} else {
					//printf("normal async in %d, size %d\n", domainIndex, pagedMemorySize/num_streams);
					cutilSafeCall( hipMemcpyAsync(d_states + local_offset, h_paged_states + local_offset,
							pagedMemorySize/num_streams, hipMemcpyHostToDevice, streams[j]) );
				}
			}
		}
		// Execute the kernel
		// Asynchronously launch num_streams kernels, each operating on its own portion of data
		for(j = 0; j < num_streams; j++) { //((timing_file!=NULL && i==0) ? 1 : 0); j < num_streams; j++){
			domainIndex = j + i*num_streams;
			if (domainIndex <= lastFullDomain) {
				local_offset = j * grid.x * threads.x ;
				if (domainIndex == lastFullDomain && (spill[1]!=0 || spill[2]!=0)) {
				    grid.x = spill[1]+1;
					solveSystem<<<grid, threads, sharedMem, streams[j]>>>(timeSteps, stepSize,
							d_states + rateStateCount*local_offset);
				} else {
					solveSystem<<<grid, threads, sharedMem, streams[j]>>>(timeSteps, stepSize,
							d_states + rateStateCount*local_offset);
				}
			}
		}

		// Asynchronoously launch num_streams memcopies
		for(j = 0; j < num_streams; j++) { //((timing_file!=NULL && i==0) ? 1 : 0); j < num_streams; j++){
			domainIndex = j + i*num_streams;
			if (domainIndex <= lastFullDomain) {
				local_offset = j * rateStateCount * grid.x * threads.x ;
				if (domainIndex == lastFullDomain && (spill[1]!=0 || spill[2]!=0)) {
					//printf("last async out %d, size %d\n", domainIndex, lastStreamMemorySize);
					cutilSafeCall( hipMemcpyAsync(h_paged_states + local_offset, d_states + local_offset,
						lastStreamMemorySize, hipMemcpyDeviceToHost, streams[j]) );
				} else {
					//printf("normal async out %d, size %d\n", domainIndex, pagedMemorySize/num_streams);
					cutilSafeCall( hipMemcpyAsync(h_paged_states + local_offset, d_states + local_offset,
						pagedMemorySize/num_streams, hipMemcpyDeviceToHost, streams[j]) );
				}
			}
		}
		
		// Execute memcpys in and out of paged memory when CUDA calls in the streams have finished
		for(j = 0; j < num_streams; j++) { //((timing_file!=NULL && i==0) ? 1 : 0); j < num_streams; j++){
			domainIndex = j + i*num_streams;
			if (domainIndex <= lastFullDomain) {
				hipStreamSynchronize(streams[j]);

				local_offset = j * rateStateCount * grid.x * threads.x ;
				global_offset = i * num_streams * grid.x * threads.x;
				
				if (domainIndex == lastFullDomain && (spill[1]!=0 || spill[2]!=0)) {
					//printf("last memcpy out %d\n", domainIndex);
					memcpy(h_states + rateStateCount * global_offset + local_offset, h_paged_states + local_offset,
							lastStreamMemorySize);
				} else {
					//printf("normal memcpy out %d\n", domainIndex);
					memcpy(h_states + rateStateCount * global_offset + local_offset, h_paged_states + local_offset,
						pagedMemorySize/num_streams);
				}

				global_offset = (i + 1) * num_streams * grid.x * threads.x;
				if (domainIndex == lastFullDomain - num_streams && (spill[1]!=0 || spill[2]!=0)) {
					//printf("last memcpy in %d\n", domainIndex);
					lastStreamMemorySize = sizeof(double)*rateStateCount*(spill[1]*threads_per_block+spill[2]);
					memcpy(h_paged_states + local_offset, h_states + rateStateCount * global_offset + local_offset,
							lastStreamMemorySize);
				} else if (domainIndex < lastFullDomain - num_streams) {
					//printf("normal memcpy in %d\n", domainIndex);
					memcpy(h_paged_states + local_offset, h_states + rateStateCount * global_offset + local_offset,
						pagedMemorySize/num_streams);
				}
			}
		}
	}

	if (timing_file) {
		// Stop global timer
		cutilCheckError(cutStopTimer(timer));

		// Calculate timing statistics
		dSeconds = cutGetTimerValue(timer)/1000.0;
		nFLOPS = (FLOPSPerTimeStep*rateStateCount + FLOPSPerFunction*FunctionEvals)*timeSteps*num_threads;
		gflops = 1.0e-9 * nFLOPS/dSeconds;

//		kernel_dSeconds = cutGetTimerValue(kernel_timer)/1000.0;
//		kernel_nFLOPS = (FLOPSPerTimeStep*rateStateCount + FLOPSPerFunction*FunctionEvals)*timeSteps*num_threads/num_streams/num_partitions;
//		kernel_gflops = 1.0e-9 * kernel_nFLOPS/kernel_dSeconds;

		// Store Stats
		fprintf(timing_file,"%s\t%s\t%d\t%d\t%d\t%d\t%d\t%f\t%f\n", cellModelName, integratorName, num_threads, num_blocks,
			threads_per_block, num_partitions, num_streams, dSeconds, gflops);
//		fprintf(timing_file,"%s\t%s\t%d\t%d\t%d\t%d\t%d\t%f\t%f\t%f\t%f\t%f\n", cellModelName, integratorName, num_threads, num_blocks,
//			threads_per_block, num_partitions, num_streams, dSeconds, gflops, kernel_dSeconds,
//			kernel_gflops, gflops/kernel_gflops*100);
	}

	// Deallocate Memory and Release Threads
	for(i = 0; i < num_streams; i++)
        	cutilSafeCall( hipStreamDestroy(streams[i]) );
	cutilSafeCall( hipFree(d_states) );
	cutilSafeCall( hipHostFree(h_paged_states) );
	hipDeviceReset();
}


////////////////////////////////////////////////////////////////////////////////
// Auxilliary Testing Functions
////////////////////////////////////////////////////////////////////////////////

void solveProblem(unsigned int timeSteps, unsigned int num_threads, unsigned int threads_per_block, 
				  unsigned int num_partitions, unsigned int num_streams, FILE *timing_file, FILE *results_file)
{
	unsigned int i, j; 
	float startTime = 0.0f;
	float endTime = 0.2f;

	double* h_states = NULL;

	h_states = (double *) safeMalloc(sizeof(double)*rateStateCount*num_threads);

	initProblem(num_threads, h_states);

	solve(h_states, startTime, endTime, timeSteps, num_threads, threads_per_block, num_partitions, num_streams, timing_file);

	if (results_file) {
		fprintf(results_file,"\n\n");

		for (i=0; i<num_threads; i++) {
			fprintf(results_file,"%d", i+1);
			for (j=0; j<rateStateCount; j++) {
				fprintf(results_file,"\t%f", h_states[i*rateStateCount+j]);
			}
			fprintf(results_file,"\n");
		}
	}

	free(h_states);
}

void startToFinish (unsigned int num_threads,unsigned int threads_per_block,unsigned int num_partitions,unsigned int num_streams,unsigned int timeSteps)
{
	//int i,j,k;
 
	FILE *file = NULL; 
	FILE *file1 = NULL;
 
	file = fopen("performance_data.txt", "rt");
	//file1 = fopen("results_data.txt", "wt");

	if (!file) {
		file = fopen("performance_data.txt", "wt");
		fprintf(file,"Cell Model\tIntegrator\tNumber of Threads\tNumber 0f Blocks\tThreads Per Block\tNumber of Partitions\tNumber of Streams\tTotal Computational Time(s)\tTotal GFLOPS\tSingle Kernel Computaional Time(s)\tKernel GFLOPS\tDevice Utilisation\n");
	} else {
		fclose(file);
		file = fopen("performance_data.txt", "at");
		if (!file) {
			fprintf(stderr, "Performance Data file could not be opened or created.");
			exit(EXIT_FAILURE);
		}
	}

	atexit( cleanup );

	//for(i=32; i<=1024; i++) if (num_threads%i==0) for(j=0; j<5; j++) solveProblem(timeSteps, num_threads, i, num_partitions, num_streams, file, file1);

	solveProblem(timeSteps, num_threads, threads_per_block, num_partitions, num_streams, file, file1);

	/*for(i=2; i<21; i+=4) {
		for(j=10; j<40; j+=4) {
			for (k=0; k<10; k++) {
				 solveProblem(timeSteps, num_threads, threads_per_block, j, i, file, file1);
			}
		}
	}*/

	if (file) fclose(file);
	if (file1) fclose(file1);
}
